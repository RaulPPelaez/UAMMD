#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2017.
  BDHI Lanczos submodule.

  Computes the mobility matrix on the fly when needed, so it is a mtrix free method.

  M·F is computed as an NBody interaction (a dense Matrix vector product).

  BdW is computed using the Lanczos algorithm [1].


  References:
  [1] Krylov subspace methods for computing hydrodynamic interactions in Brownian dynamics simulations.
  -http://dx.doi.org/10.1063/1.4742347
  [2] J. Chem. Phys. 137, 064106 (2012); doi: 10.1063/1.4742347

*/
#include"BDHI_Lanczos.cuh"
#include "misc/LanczosAlgorithm/MatrixDot.h"
#include"Interactor/NBody.cuh"
#include "utils/container.h"


namespace uammd{
  namespace BDHI{

    Lanczos::Lanczos(shared_ptr<ParticleGroup> pg, Parameters par):
      pg(pg),
      hydrodynamicRadius(par.hydrodynamicRadius),
      temperature(par.temperature),
      tolerance(par.tolerance),
      rpy(par.viscosity),par(par){
      System::log<System::MESSAGE>("[BDHI::Lanczos] Initialized");
      auto pd = pg->getParticleData();
      //Lanczos algorithm computes,
      //given an object that computes the product of a Matrix(M) and a vector(v), sqrt(M)·v
      lanczosAlgorithm = std::make_shared<lanczos::Solver>();
      if(par.hydrodynamicRadius>0)
	System::log<System::MESSAGE>("[BDHI::Lanczos] Self mobility: %g", rpy(0,par.hydrodynamicRadius, par.hydrodynamicRadius).x);
      else{
	System::log<System::MESSAGE>("[BDHI::Lanczos] Self mobility dependent on particle radius as 1/(6πηa)");
      }
      if(par.hydrodynamicRadius<0 and ! pd->isRadiusAllocated())
	System::log<System::CRITICAL>("[BDHI::Lanczos] You need to provide Lanczos with either an hydrodynamic radius or via the individual particle radius.");
      if(par.hydrodynamicRadius>0 and pd->isRadiusAllocated())
	System::log<System::MESSAGE>("[BDHI::Lanczos] Taking particle radius from parameter's hydrodynamicRadius");
      //Init rng
      hiprandCreateGenerator(&curng, HIPRAND_RNG_PSEUDO_DEFAULT);
      hiprandSetPseudoRandomGeneratorSeed(curng, pd->getSystem()->rng().next());
      thrust::device_vector<real> noise(30000);
      auto noise_ptr = thrust::raw_pointer_cast(noise.data());
      //Warm cuRNG
      curandgeneratenormal(curng, noise_ptr, noise.size(), 0.0, 1.0);
      curandgeneratenormal(curng, noise_ptr, noise.size(), 0.0, 1.0);
    }

    namespace Lanczos_ns{
      /*Compute the product Mv = M·v, computing M on the fly when needed, without storing it*/
      /*This critital compute is the 99% of the execution time in a BDHI simulation*/
      /*Each thread handles one particle with the other N, including itself*/
      /*That is 3 full lines of M, or 3 elements of M·v per thread, being the x y z of ij with j=0:N-1*/
      /*In other words. M is made of NxN boxes of size 3x3,
	defining the x,y,z mobility between particle pairs,
	each thread handles a row of boxes and multiplies it by three elements of v*/
      /*vtype can be real3 or real4*/
      template<class vtype>
      struct NbodyMatrixFreeMobilityDot{
	typedef real3 computeType;
	typedef real4 infoType; //v[i], radius[i]
	NbodyMatrixFreeMobilityDot(vtype* v,
				   real3 *Mv,
				   real rh, //Used only if radius is null
				   real * radius,
				   BDHI::RotnePragerYamakawa rpy):
	  v(v), Mv(Mv), rpy(rpy), radius(radius), rh(rh){}
	/*Start with 0*/
	inline __device__ computeType zero(){ return make_real3(0);}

	inline __device__ infoType getInfo(int pi){
	  return make_real4(make_real3(v[pi]), radius?radius[pi]:rh);
	}
	/*Just count the interaction*/
	inline __device__ computeType compute(const real4 &pi, const real4 &pj,
					      const infoType &info_i, const infoType &info_j){
	  /*Distance between the pair*/
	  const real3 rij = make_real3(pi)-make_real3(pj);
	  const real r = sqrt(dot(rij, rij));
	  const real3 vj = make_real3(info_j);
	  /*Compute RPY coefficients, see more info in BDHI::RPYutils::RPY*/
	  const real2 c12 = rpy(r, info_i.w, info_j.w);

	  const real f = c12.x;
	  const real gdivr2 = c12.y;

	  /*Self mobility*/
	  if(r==real(0.0))
	    return f*vj;
	  /*This expression is a little obfuscated, Mij*vj = f(rij)·I + g(rij)/rij^2 · \vec{rij}\diadic \vec{rij} ) · \vec{vij}
	    Where f and g are the hydrodinamic kernel coefficients
	  */
	  const real gv = gdivr2*dot(rij, vj);
	  /*gv = g(r)·( vx·rx + vy·ry + vz·rz )*/
	  /*(g(r)·v·(r(diadic)r) )_ß = gv·r_ß*/
	  const real3 Mv_t = f*vj + gv*rij;
	  return Mv_t;
	}
	/*Sum the result of each interaction*/
	inline __device__ void accumulate(computeType &total, const computeType &cur){total += cur;}

	/*Write the final result to global memory*/
	inline __device__ void set(int id, const computeType &total){
	  Mv[id] = total;
	}
	vtype* v;
	real3* Mv;
	real rh;
	real *radius;
	BDHI::RotnePragerYamakawa rpy;
      };

      /*A functor to pass to LanczosAlgorithm the operation Mv = M·v*/
      template<typename vtype>
      struct Dotctor: public lanczos::MatrixDot{
	using myTransverser = Lanczos_ns::NbodyMatrixFreeMobilityDot<vtype>;
	myTransverser Mv_tr;
    	shared_ptr<NBody> nbody;
	hipStream_t st;
	Dotctor(BDHI::RotnePragerYamakawa rpy, real rh, real *radius,
		shared_ptr<NBody> nbody, hipStream_t st):
	  Mv_tr(nullptr, nullptr, rh, radius, rpy),
	  nbody(nbody),
	  st(st)
	  {}

	inline void operator()(real* v, real *Mv){
	  Mv_tr.v  = (vtype*)v; /*src*/
	  Mv_tr.Mv = (real3*)Mv; /*Result*/
	  nbody->transverse(Mv_tr, st);
	}

      };
    }

    void Lanczos::computeMF(real3* MF, hipStream_t st){
      /*For M·v product. Being M the Mobility and v an arbitrary array.
	The M·v product can be seen as an Nbody interaction Mv_j = sum_i(Mij*vi)
	Where Mij = RPY( |rij|^2 ).

	Although M is 3Nx3N, it is treated as a Matrix of NxN boxes of size 3x3,
	and v is a vector3.
      */
      System::log<System::DEBUG1>("[BDHI::Lanczos] MF");
      using myTransverser = Lanczos_ns::NbodyMatrixFreeMobilityDot<real4>;
      auto pd = pg->getParticleData();
      auto force = pd->getForce(access::location::gpu, access::mode::read);
      auto radius = pd->getRadiusIfAllocated(access::location::gpu, access::mode::read);
      real * radius_ptr =  this->hydrodynamicRadius>0?nullptr:radius.raw();
      myTransverser Mv_tr(force.raw(), MF, this->hydrodynamicRadius, radius_ptr, rpy);
      NBody nbody(pg);
      nbody.transverse(Mv_tr, st);
    }

    void Lanczos::computeBdW(real3 *BdW, hipStream_t st){
      System::log<System::DEBUG1>("[BDHI::Lanczos] BdW");
      if(temperature > real(0.0)){
	st = 0;
	int numberParticles = pg->getNumberParticles();
	auto nbody = std::make_shared<NBody>(pg);
	auto pd = pg->getParticleData();
	/*Lanczos Algorithm needs a functor that provides the dot product of M and a vector*/
	auto radius = pd->getRadiusIfAllocated(access::location::gpu, access::mode::read);
	real * radius_ptr =  this->hydrodynamicRadius>0?nullptr:radius.raw();
	Lanczos_ns::Dotctor<real3> Mdot(rpy, this->hydrodynamicRadius, radius_ptr, nbody, st);
	//Filling V instead of an external array (for v in sqrt(M)·v) is faster
	uninitialized_cached_vector<real3> noise(numberParticles);
	curandgeneratenormal(curng, (real*)noise.data().get(),
			     3*numberParticles + (3*numberParticles)%2,
			     real(0.0), real(1.0));
	//lanczosAlgorithm->solve(Mdot, (real*) BdW, noise, numberParticles, st);
	lanczosAlgorithm->run(Mdot, (real*) BdW, (real*)noise.data().get(),
			      tolerance, 3*numberParticles, st);
      }
    }
  }
}
