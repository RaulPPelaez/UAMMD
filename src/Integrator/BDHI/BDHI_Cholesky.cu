#include "hip/hip_runtime.h"
/*Raul P. Pelaez. 2017. Cholesky BDHI submodule implementation

  Computes the hydrodynamic interactions between particles in the system by
  maintaining a 3Nx3N mobility matrix with the RPY tensor in memory
  and explicitly computing M·F as a matrix-vector product.

  Note that only the upper part of M is stored, as M is symmetric.

  The brownian noise is computed as BdW = chol(M)·dW with cuSOLVER and cuBLAS

References:
[1] https://github.com/RaulPPelaez/UAMMD/wiki/BDHI_Cholesky
[2] https://github.com/RaulPPelaez/UAMMD/wiki/NBody-Forces
*/
#include"BDHI_Cholesky.cuh"
#include"Interactor/NBody.cuh"
#include"utils/debugTools.h"
#include<hipblas.h>
#include"utils/cublasDebug.h"
#include<hipsolver.h>
#include"utils/cusolverDebug.h"

#include<utils/cuda_lib_defines.h>

#include<fstream>
namespace uammd{
  namespace BDHI{

    namespace Cholesky_ns{
      template<class IndexIter>
      /*Fills the 3Nx3N Mobility matrix with Mij = RPY(|rij|)(I-r^r) in blocks of 3x3 matrices*/
      __global__ void fillMobilityRPYD(real * __restrict__ M,
				       const  real4* __restrict__ R,
				       IndexIter indexIter,
				       uint N,
				       real hydrodynamicRadius, //Used if radius is null
				       real *radius,
				       BDHI::RotnePragerYamakawa rpy){
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id>=N) return;
	int i = indexIter[id];
	uint n = 3*N;

	/*Self Diffusion*/
	for(int k = 0; k < 3; k++)
	  for(int l = 0; l < 3; l++){
	    M[3*id + k + n*(3*id + l)] =  real(0.0);
	  }

	const real radius_i = radius?radius[i]:hydrodynamicRadius;
	real M0 = rpy(0, radius_i, radius_i).x;
	M[3*id + 0 + n*(3*id + 0)] = M0;
	M[3*id + 1 + n*(3*id + 1)] = M0;
	M[3*id + 2 + n*(3*id + 2)] = M0;

	real3 rij;
	real* rijp = &(rij.x);
	real3 ri = make_real3(R[i]);
	for(int j=id+1; j<N; j++){
	  const int global_j = indexIter[j];
	  const real radius_j = radius?radius[global_j]:hydrodynamicRadius;
	  rij = make_real3(R[global_j]) - ri;

	  const real r = sqrt(dot(rij, rij));
	  /*Rotne-Prager-Yamakawa tensor: RPY = f(r)*I + g(r)*r\diadic r*/

	  const real2 c12 = rpy(r, radius_i, radius_j);
	  const real c1 = c12.x;
	  const real c2 = c12.y;
	  /*3x3 Matrix for each particle pair*/
	  for(int k = 0; k < 3; k++)
	    for(int l = 0; l < 3; l++)
	      M[3*id + k + n*(3*j + l)] = c2*rijp[k]*rijp[l];
	  /*Diagonal*/
	  for(int k = 0; k<3; k++)  M[3*id + k + n*(3*j + k)] += c1;
	}
      }
    }

    Cholesky::Cholesky(shared_ptr<ParticleGroup> pg, Parameters par):
      pg(pg),
      par(par),
      rpy(par.viscosity){
      System::log<System::MESSAGE>("[BDHI::Cholesky] Initialized");
      int numberParticles = pg->getNumberParticles();
      force3.resize(numberParticles, real3());
      mobilityMatrix.resize(pow(3*numberParticles,2)+1, real());
      auto pd = pg->getParticleData();
      if(par.hydrodynamicRadius>0)
	System::log<System::MESSAGE>("[BDHI::Cholesky] Self mobility: %g", rpy(0,par.hydrodynamicRadius, par.hydrodynamicRadius).x);
      else{
	System::log<System::MESSAGE>("[BDHI::Cholesky] Self mobility dependent on particle radius as 1/(6πηa)");
      }
      if(par.hydrodynamicRadius<0 and ! pd->isRadiusAllocated())
	System::log<System::CRITICAL>("[BDHI::Cholesky] You need to provide Cholesky with either an hydrodynamic radius or via the individual particle radius.");
      if(par.hydrodynamicRadius>0 and pd->isRadiusAllocated())
	System::log<System::MESSAGE>("[BDHI::Cholesky] Taking particle radius from parameter's hydrodynamicRadius");
      /*Init cuSolver for BdW*/
      CusolverSafeCall(hipsolverDnCreate(&solver_handle));
      h_work_size = 0;//work size of operation
      auto d_M = thrust::raw_pointer_cast(mobilityMatrix.data());
      CusolverSafeCall(cusolverDnpotrf_bufferSize(solver_handle,
						  HIPBLAS_FILL_MODE_UPPER,
						  3*numberParticles,
						  d_M, 3*numberParticles,
						  &h_work_size));
      CudaSafeCall(hipMalloc(&d_work, h_work_size));
      CudaSafeCall(hipMalloc(&d_info, sizeof(int)));
      /*Init cuBLAS for MF*/
      CublasSafeCall(hipblasCreate(&handle));
      /*Create noise*/
      hiprandCreateGenerator(&curng, HIPRAND_RNG_PSEUDO_DEFAULT);
      hiprandSetPseudoRandomGeneratorSeed(curng, pd->getSystem()->rng().next());
      /*Create a temporal vector to warm up hiprand*/
      thrust::device_vector<real> noise(numberParticles+1);
      //Curand fill with gaussian numbers with mean 0 and var 1
      /*This shit is obscure, hiprand will only work with an even number of elements*/
      auto d_noise = thrust::raw_pointer_cast(noise.data());
      curandgeneratenormal(curng, d_noise, 3*numberParticles + ((3*numberParticles)%2), real(0.0), real(1.0));
      isMup2date = false;
    }

    Cholesky::~Cholesky(){
      hipblasDestroy(handle);
      hiprandDestroyGenerator(curng);
      hipFree(d_work);
      hipFree(d_info);
    }

    void Cholesky::init(){}

    void Cholesky::setup_step(hipStream_t st){
      System::log<System::DEBUG3>("[BDHI::Cholesky] Setup Step");
      int numberParticles = pg->getNumberParticles();
      auto pd = pg->getParticleData();
      auto pos = pd->getPos(access::location::gpu, access::mode::read);
      auto groupIter = pg->getIndexIterator(access::location::gpu);
      auto d_M = thrust::raw_pointer_cast(mobilityMatrix.data());
      int BLOCKSIZE = 128;
      int Nthreads = BLOCKSIZE<numberParticles?BLOCKSIZE:numberParticles;
      int Nblocks=numberParticles/Nthreads + ((numberParticles%Nthreads)?1:0);
      auto radius = pd->getRadiusIfAllocated(access::location::gpu, access::mode::read);
      /*Fill the upper part of symmetric mobility matrix*/
      Cholesky_ns::fillMobilityRPYD<<<Nblocks, Nthreads, 0 ,st>>>(d_M,
       								  pos.raw(),
       								  groupIter,
       								  numberParticles,
								  par.hydrodynamicRadius,
								  radius.raw(),
       								  rpy);
      /*M contains the mobility tensor in this step*/
      isMup2date = true;
    }

    namespace Cholesky_ns{
      template<class IndexIter, class Real3OutputIterator>
      __global__ void real4ToReal3(IndexIter indexIter,
				   real4 * in,
				   Real3OutputIterator out,
				   int N){
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id>=N) return;
	int i = indexIter[id];
	out[id] = make_real3(in[i]);
      }

    }

    void Cholesky::computeMF(real3* MF, hipStream_t st){
      System::log<System::DEBUG3>("[BDHI::Cholesky] MF");
      /*computeMF should be called before computeBdW*/
      static bool warning_printed = false;
      if(!isMup2date){
	setup_step(st);
	if(!warning_printed){
	  System::log<System::WARNING>("[BDHI::Cholesky] You should call computeMF inmediatly after setup_step, otherwise M will be compute twice per step!");
	  warning_printed = true;
	}
      }

      int numberParticles = pg->getNumberParticles();
      /*Morphs a real4 vector into a real3 one, needed by cublas*/
      CublasSafeCall(hipblasSetStream(handle, st));
      auto pd = pg->getParticleData();
      auto force = pd->getForce(access::location::gpu, access::mode::read);
      auto indexIter = pg->getIndexIterator(access::location::gpu);
      int BLOCKSIZE = 128;
      int Nthreads = BLOCKSIZE<numberParticles?BLOCKSIZE:numberParticles;
      int Nblocks=numberParticles/Nthreads + ((numberParticles%Nthreads)?1:0);
      Cholesky_ns::real4ToReal3<<<Nblocks, Nthreads, 0 , st>>>(indexIter,
							       force.raw(),
							       force3.begin(),
							       numberParticles);
      real alpha = 1.0;
      real beta = 0;
      /*Compute M·F*/
      real * d_M = thrust::raw_pointer_cast(mobilityMatrix.data());
      real * d_force3 = (real*)thrust::raw_pointer_cast(force3.data());
      CublasSafeCall(cublassymv(handle, HIPBLAS_FILL_MODE_UPPER,
				3*numberParticles,
				&alpha,
				d_M, 3*numberParticles,
				d_force3, 1,
				&beta,
				(real *)MF, 1));
    }


    void Cholesky::computeBdW(real3 *BdW, hipStream_t st){
      System::log<System::DEBUG3>("[BDHI::Cholesky] BdW");
      if(!isMup2date) setup_step();
      /*computeBdw messes up M, fills it with B*/
      isMup2date = false;
      int numberParticles = pg->getNumberParticles();
      CusolverSafeCall(hipsolverSetStream(solver_handle, st));
      real * d_M = thrust::raw_pointer_cast(mobilityMatrix.data());
      /*Perform cholesky factorization, store B on LOWER part of M matrix*/
      CusolverSafeCall(cusolverDnpotrf(solver_handle, HIPBLAS_FILL_MODE_UPPER,
				       3*numberParticles, d_M, 3*numberParticles, d_work, h_work_size, d_info));
      hiprandSetStream(curng, st);
      /*Gen new noise in BdW*/
      curandgeneratenormal(curng,
			   (real*) BdW,
			   3*numberParticles + ((3*numberParticles)%2),
			   real(0.0), real(1.0));
      CublasSafeCall(hipblasSetStream(handle, st));
      /*Compute B·dW -> y = M·y*/
      CublasSafeCall(cublastrmv(handle, //B is an upper triangular matrix (with non unit diagonal)
				HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
				3*numberParticles,
				d_M, 3*numberParticles,
				(real*)BdW, 1));
    }


    // namespace Cholesky_ns{
    //   /*Exactly the same as Lanczos_ns::divMTranverser.
    // 	It is placed here for convinience when performing tests that involve
    // 	changing the input parameters to the class*/
    //   /*This Nbody Transverser computes the analytic divergence of the RPY tensor*/
    //   // https://github.com/RaulPPelaez/UAMMD/wiki/Nbody-Forces
    //   // https://github.com/RaulPPelaez/UAMMD/wiki/Transverser
    //   struct divMTransverser{
    // 	divMTransverser(real3* divM, real rh, real* radius): divM(divM), rh(rh), radius(radius){}

    // 	real getInfo(int i){
    // 	  return radius?radius[i]:rh;
    // 	}
    // 	inline __device__ real3 zero(){return make_real3(real(0.0));}
    // 	inline __device__ real3 compute(const real4 &pi, const real4 &pj, const real &a_i, const real &a_j){
    // 	  /*Work in units of rh*/
    // 	  const real3 r12 = (make_real3(pi)-make_real3(pj))*invrh;
    // 	  const real r2 = dot(r12, r12);
    // 	  if(r2==real(0.0))
    // 	    return make_real3(real(0.0));
    // 	  real invr = rsqrt(r2);
    // 	  /*Just the divergence of the RPY tensor in 2D, taken from A. Donev's notes*/
    // 	  /*The 1/6pia is in M0, the factor kT is in the integrator, and the factor 1/a is in set*/
    // 	  if(r2>real(4.0)){
    // 	    real invr2 = invr*invr;
    // 	    return real(0.75)*(r2-real(2.0))*invr2*invr2*r12*invr;
    // 	  }
    // 	  else{
    // 	    return real(0.09375)*r12*invr;
    // 	  }
    // 	}
    // 	inline __device__ void accumulate(real3 &total, const real3 &cur){total += cur;}

    // 	inline __device__ void set(int id, const real3 &total){
    // 	  divM[id] = M0*total*invrh;
    // 	}
    //   private:
    // 	real3* divM;
    // 	real M0;
    // 	real rh, invrh;
    //   };

    // }

    // void Cholesky::computeDivM(real3* divM, hipStream_t st){
    //   /*A simple NBody transverser, see https://github.com/RaulPPelaez/UAMMD/wiki/NBody-Forces */
    //   Cholesky_ns::divMTransverser divMtr(divM, selfMobility, par.hydrodynamicRadius);
    //   NBody nbody_divM(pd, pg, sys);

    //   nbody_divM.transverse(divMtr, st);
    // }
  }
}
