#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2016. Brownian Euler Maruyama with hydrodynamics Integrator derived class implementation

  Solves the following stochastich differential equation:
  X[t+dt] = dt(K·X[t]+M·F[t]) + sqrt(2*kb*T*dt)·B·dW
  Being:
  X - Positions
  M - Mobility matrix -> M = D/kT
  K - Shear matrix
  dW- Brownian noise vector
  B - B*B^T = M -> i.e Cholesky decomposition B=chol(M) or Square root B=sqrt(M)
  divM - Divergence of the mobility matrix, zero in 3D and 2D, but non zero in q2D, which is turned off for the moment.

  The Mobility matrix is computed via the Rotne Prager Yamakawa tensor.

  The module offers several ways to compute and solve the different terms.

  BDHI::Cholesky:
  -Computing M·F and B·dW  explicitly storing M and performing a Cholesky decomposition on M.

  BDHI::Lanczos:
  -A Lanczos iterative method to reduce M to a smaller Krylov subspace and performing the operation B·dW there, the product M·F is performed in a matrix-free way, recomputing M every time M·v is needed.

  BDHI::PSE:
  -The Positively Split Edwald Method, which takes the computation to fourier space. [2]

  REFERENCES:

  1- Krylov subspace methods for computing hydrodynamic interactions in Brownian dynamics simulations
  J. Chem. Phys. 137, 064106 (2012); doi: 10.1063/1.4742347
  2- Rapid sampling of stochastic displacements in Brownian dynamics simulations
  The Journal of Chemical Physics 146, 124116 (2017); doi: http://dx.doi.org/10.1063/1.4978242

  TODO:
  100- Optimize streams
*/
#include"BDHI_EulerMaruyama.cuh"

namespace uammd{
  namespace BDHI{
    template<class Method>
    EulerMaruyama<Method>::EulerMaruyama(shared_ptr<ParticleData> pd,
					 shared_ptr<ParticleGroup> pg,
					 shared_ptr<System> sys,
					 Parameters par):
      Integrator(pd, pg, sys, "BDHI::EulerMaruyama/"+type_name<Method>()),
      K(par.K),
      par(par),
      steps(0)
    {
      bdhi = std::make_shared<Method>(pd, pg, sys, par);
      sys->log<System::MESSAGE>("[BDHI::EulerMaruyama] Initialized");

      int numberParticles = pg->getNumberParticles();

      sys->log<System::MESSAGE>("[BDHI::EulerMaruyama] Temperature: %f", par.temperature);
      sys->log<System::MESSAGE>("[BDHI::EulerMaruyama] Viscosity: %f", par.viscosity);
      sys->log<System::MESSAGE>("[BDHI::EulerMaruyama] Time step: %f", par.dt);
      if(par.hydrodynamicRadius>0)
	sys->log<System::MESSAGE>("[BDHI::EulerMaruyama] Hydrodynamic Radius: %f", par.hydrodynamicRadius);
      if(par.K.size()==3){
	real3 Kx = par.K[0];
	real3 Ky = par.K[1];
	real3 Kz = par.K[2];
	sys->log<System::MESSAGE>("[BDHI::EulerMaruyama] Shear Matrix: [ %f %f %f; %f %f %f; %f %f %f ]",
				  Kx.x, Kx.y, Kx.z,
				  Ky.x, Ky.y, Ky.z,
				  Kz.x, Kz.y, Kz.z);
      }

      hipStreamCreate(&stream);
      hipStreamCreate(&stream2);

      /*Result of multiplyinf M·F*/
      MF.resize(numberParticles, real3());
      BdW.resize(numberParticles+1, real3());
      //if(par.is2D) divM.resize(numberParticles, real3());

    }
    template<class Method>
    EulerMaruyama<Method>::~EulerMaruyama(){
      sys->log<System::MESSAGE>("[BDHI::EulerMaruyama] Destroyed");
      hipStreamDestroy(stream);
      hipStreamDestroy(stream2);
    }

    namespace EulerMaruyama_ns{
      /*
	dR = dt(KR+MF) + sqrt(2*T*dt)·BdW +T·divM·dt -> divergence is commented out for the moment
      */
      /*With all the terms computed, update the positions*/
      /*T=0 case is templated*/
      template<class IndexIterator>
      __global__ void integrateGPUD(real4* __restrict__ pos,
				    IndexIterator indexIterator,
				    const real3* __restrict__ MF,
				    const real3* __restrict__ BdW,
				    const real3* __restrict__ K,
				    //const real3* __restrict__ divM,
				    int N,
				    real sqrt2Tdt, real T, real dt, bool is2D){
	uint id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=N) return;
	int i = indexIterator[id];
	/*Position and color*/
	real4 pc = pos[i];
	real3 p = make_real3(pc);
	real c = pc.w;

	/*Shear stress*/
	if(K){
	  real3 KR = make_real3(0);
	  KR.x = dot(K[0], p);
	  KR.y = dot(K[1], p);
	  /*2D clause. Although K[2] should be 0 in 2D anyway...*/
	  if(!is2D)
	    KR.z = dot(K[2], p);
	  p += KR*dt;
	}
	/*Update the position*/
	p += MF[id]*dt;
	/*T=0 is treated specially, there is no need to produce noise*/
	if(BdW){
	  real3 bdw  = BdW[id];
	  if(is2D)
	    bdw.z = 0;
	  p += sqrt2Tdt*bdw;
	}
	/*If we are in q2D and the divergence term exists*/
	// if(divM){
	//   real3 divm = divM[id];
	//   //divm.z = real(0.0);
	//   //p += params.T*divm*params.invDelta*params.invDelta*params.dt; //For RFD
	//   p += T*dt*divm;
	// }
	/*Write to global memory*/
	pos[i] = make_real4(p,c);
      }
    }


    /*Advance the simulation one time step*/
    template<class Method>
    void EulerMaruyama<Method>::forwardTime(){
      sys->log<System::DEBUG1>("[BDHI::EulerMaruyama] Performing integration step %d", steps);
      /*
	dR = dt(KR+MF) + sqrt(2*T*dt)·BdW +T·divM·dt
      */
      steps++;

      for(auto forceComp: interactors) forceComp->updateSimulationTime(steps*par.dt);

      if(steps==1){
	for(auto forceComp: interactors){
	  forceComp->updateTimeStep(par.dt);
	  forceComp->updateTemperature(par.temperature);
	  forceComp->updateBox(par.box);
	}
      }

      int numberParticles = pg->getNumberParticles();

      int BLOCKSIZE = 128; /*threads per block*/
      int nthreads = BLOCKSIZE<numberParticles?BLOCKSIZE:numberParticles;
      int nblocks = numberParticles/nthreads +  ((numberParticles%nthreads!=0)?1:0);

      auto indexIter = pg->getIndexIterator(access::location::gpu);
      {
	auto force = pd->getForce(access::location::gpu, access::mode::write);
	/*Reset force*/
	fillWithGPU<<<nblocks, nthreads, 0, stream>>>(force.raw(),
						     indexIter, make_real4(0), numberParticles);
      }
      /*Compute new force*/
      for(auto forceComp: interactors) forceComp->sumForce(stream);

      bdhi->setup_step(stream);

      auto d_MF = thrust::raw_pointer_cast(MF.data());
      bdhi->computeMF(d_MF, stream);


      if(par.temperature>0){
	auto d_BdW = thrust::raw_pointer_cast(BdW.data());
	bdhi->computeBdW(d_BdW, stream);
      }

      // if(par.is2D){
      // 	auto d_divM = thrust::raw_pointer_cast(divM.data());
      // 	bdhi->computeDivM(d_divM, stream2);
      // }

      real sqrt2Tdt = sqrt(2*par.dt*par.temperature);

      bdhi->finish_step(stream);

      /*Update the positions*/
      /* R += KR + MF + sqrt(2dtT)BdW + kTdivM*/

      real3* d_BdW = nullptr;
      if(par.temperature > 0) d_BdW = thrust::raw_pointer_cast(BdW.data());

      real3* d_K = nullptr;
      if(par.K.size() > 0) d_K = thrust::raw_pointer_cast(K.data());

      //real3* d_divM = nullptr;
      //if(par.is2D) d_divM = thrust::raw_pointer_cast(divM.data());

      auto pos = pd->getPos(access::location::gpu, access::mode::readwrite);

      //hipStreamSynchronize(stream2);

      EulerMaruyama_ns::integrateGPUD<<<nblocks, nthreads, 0, stream>>>(pos.raw(),
									indexIter,
									d_MF,
									d_BdW,
									d_K,
									//d_divM,
									numberParticles,
									sqrt2Tdt,
									par.temperature,
									par.dt, par.is2D);


    }
    template<class Method>
    real EulerMaruyama<Method>::sumEnergy(){
      //Sum 1.5*kT to each particle
      auto energy = pd->getEnergy(access::gpu, access::readwrite);
      auto energy_gr = pg->getPropertyIterator(energy);
      auto energy_per_particle = thrust::make_constant_iterator<real>(1.5*par.temperature);
      thrust::transform(thrust::cuda::par,
			energy_gr, energy_gr + pg->getNumberParticles(),
			energy_per_particle,
			energy_gr,
			thrust::plus<real>());
      return 0;
    }
  }
}
