#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2017-2021. PairForces definition.

  PairForces Module is an interactor that computes forces and/or energies
  between pairs of particle closer to a given cut off distance. If the cut off
  reaches a certain threshold the algorithm switches to n-body.

  A Potential describing the interaction must be provided.
  See misc/Potential.cuh and https://github.com/RaulPPelaez/UAMMD/wiki/Potential
  for more info on potentials and how to implement them.

  See https://github.com/RaulPPelaez/UAMMD/wiki/Pair-Forces for more info.
*/

#include "PairForces.cuh"
#include "Potential/PotentialUtils.cuh"
namespace uammd {

template <class MyPotential, class NL>
PairForces<MyPotential, NL>::PairForces(shared_ptr<ParticleGroup> pg,
                                        Parameters par,
                                        shared_ptr<MyPotential> pot)
    : Interactor(
          pg,
          "PairForces/" +
              stringUtils::removePattern(type_name<NL>(), "uammd::") + "/" +
              stringUtils::removePattern(type_name<MyPotential>(), "uammd::")),
      box(par.box), pot(pot), nl(par.nl), nb(nullptr) {
  constexpr bool hasTransverser =
      Potential::has_getTransverser<MyPotential>::value;
  if (not hasTransverser) {
    auto potname =
        stringUtils::removePattern(type_name<MyPotential>(), "uammd::");
    sys->log<System::ERROR>("[PairForces] No valid Transverser found in %s.",
                            potname.c_str());
    sys->log<System::ERROR>("[PairForces] A member function called "
                            "getTransverser must be defined:");
  }
  sys->log<System::MESSAGE>("[PairForces] Using Box with size: %g %g %g",
                            box.boxSize.x, box.boxSize.y, box.boxSize.z);
  this->setDelegate(pot.get());
}

template <class MyPotential, class NL>
template <class Transverser>
void PairForces<MyPotential, NL>::sumTransverser(Transverser &tr,
                                                 hipStream_t st) {
  this->rcut = pot->getCutOff();
  sys->log<System::DEBUG3>("[PairForces] Using cutOff: %f", this->rcut);
  bool useNeighbourList = true;
  if (box.boxSize.x <= 3 * rcut and box.boxSize.y <= 3 * rcut and
      box.boxSize.z <= 3 * rcut) {
    useNeighbourList = false;
  }
  if (useNeighbourList) {
    if (!nl) {
      nl = std::make_shared<NL>(pg);
    }
    nl->update(box, rcut, st);
    sys->log<System::DEBUG2>("[PairForces] Transversing neighbour list");
    nl->transverseList(tr, st);
  } else {
    if (!nb) {
      nb = std::make_shared<NBody>(pg);
    }
    sys->log<System::DEBUG2>("[PairForces] Transversing NBody");
    nb->transverse(tr, st);
  }
}

template <class MyPotential, class NL>
void PairForces<MyPotential, NL>::sum(Computables comp, hipStream_t st) {
  // Try to use getForceTransverser, if not present try to use
  // getForceEnergyTransverser, if also not present assume zero force
  sys->log<System::DEBUG1>("[PairForces] Summing interaction");
  auto ft =
      Potential::getIfHasTransverser<MyPotential>::get(pot, comp, box, pd);
  this->sumTransverser(ft, st);
}
} // namespace uammd
