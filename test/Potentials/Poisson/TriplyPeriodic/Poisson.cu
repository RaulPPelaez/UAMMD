#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2019-2022. Poisson test
Computes the electric field between two opposite charges placed in a periodic box.
USAGE:
./poisson [gw] [L] [r]

gw: Gaussian width of the charges
L: Box size (cubic box)
r: distance between the charges

In the limit when L->inf the field between the two particles should be:
Ex =  exp(-r**2/(4.0*gw**2))/(4*pi**1.5*gw*r) - erf(r/(2.0*gw))/(4*pi*r**2);
*/
#include"uammd.cuh"
#include"Interactor/SpectralEwaldPoisson.cuh"
#include<fstream>
#include<iomanip>
using namespace uammd;
using std::make_shared;
using std::endl;

int main(int argc, char *argv[]){
  int N = 2;
  real L = std::stod(argv[1]);
  real r = std::stod(argv[2]);
  real tolerance = std::stod(argv[3]);
  real gw = std::stod(argv[4]);
  real split = std::stod(argv[5]);
  auto sys = make_shared<System>(argc, argv);
  auto pd = make_shared<ParticleData>(N, sys);
  Box box(L);
  {
    auto pos = pd->getPos(access::location::cpu, access::mode::write);
    auto charge = pd->getCharge(access::location::cpu, access::mode::write);
    auto ori = make_real4(make_real3(sys->rng().uniform3(-0.5, 0.5))*L, 0);
    pos[0] = make_real4(-r*0.5,0,0,0)+ori;
    pos[1] = make_real4( r*0.5,0,0,0)+ori;
    charge[0] = 1;
    charge[1] = -1;
  }
  Poisson::Parameters par;
  par.box = box;
  par.epsilon = 1;
  par.gw = gw;
  par.tolerance = tolerance;
  par.split = split;
  //par.upsampling = 1.0;
  auto poisson = make_shared<Poisson>(pd, par);
  {
    auto force = pd->getForce(access::location::gpu, access::mode::write);
    thrust::fill(thrust::cuda::par, force.begin(), force.end(), real4());
    auto energy = pd->getEnergy(access::location::gpu, access::mode::write);
    thrust::fill(thrust::cuda::par, energy.begin(), energy.end(), real());
  }
  poisson->sum({.force=true, .energy=true, .virial=false});
  {
    auto pos = pd->getPos(access::location::cpu, access::mode::read);
    auto force = pd->getForce(access::location::cpu, access::mode::read);
    auto energy = pd->getEnergy(access::location::cpu, access::mode::read);
    auto charge = pd->getCharge(access::location::cpu, access::mode::read);
    real3 p;
    fori(0,N){
      real4 pc = pos[i];
      p = make_real3(pc);
      int type = charge[i];
      std::cout<<std::setprecision(15)<<p<<" q: "<<charge[i]<<" F: "<<force[i]<<" E: "<<energy[i]<<endl;
    }
  }
  sys->finish();
  return 0;
}
