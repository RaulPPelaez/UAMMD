#include "hip/hip_runtime.h"
/* Raul P. Pelaez 2021
   More about System.
   Besides its logging capabilities, System also provides a random number generator that can be used to seed other generators
   throughout the code.
   We will also see other things that System can do.
 */

//uammd.cuh is the basic uammd include containing, among other things, the System struct.
#include<uammd.cuh>
#include<random>
#include<thrust/device_vector.h>
using namespace uammd;

int main(int argc, char* argv[]){
  //Initialize System
  auto sys = std::make_shared<System>(argc, argv);
  //We can access the System's random number generator via sys->rng()
  //The first thing we should do is seed it with something controlled by us.
  //Since UAMMD modules will use this generator when in need of a seed, this will allow us to have deterministic runs.
  //Meaning that the same UAMMD code ran with the same seed will result in the same random numbers.
  //Besides numerical error, coming from the non-sequential nature of GPU execution, this gives reproducible runs.
  //If not seeded, System will initialize the generator using the current number of seconds since epoch.
  auto seed = 0x12345;
  //Alternatively, we could use C++'s random_device to seed differently each run:
  //std::random_device rd;
  //auto seed = rd();
  sys->rng().setSeed(seed);
  //This rng can generate numbers in several ways, lets see a few:
  //A number chosen among all the representable by uint (0 and 2^32-1)
  uint integer = sys->rng().next32();
  sys->log<System::MESSAGE>("An integer number between 0 and 2^32-1: %u", integer);
  //A number chosen among all the representable by uint64_t  (0 and 2^64-1)
  uint64_t long_integer  = sys->rng().next();
  sys->log<System::MESSAGE>("An integer number between 0 and 2^64-1: %lu", long_integer);
  //An uniform random number between 0 and1
  double uniform = sys->rng().uniform(0, 1);
  sys->log<System::MESSAGE>("An uniformly distributed number between 0 and 1: %.13g", uniform);
  //A normally distributed number with 0 mean and standard deviation 1
  double normal = sys->rng().gaussian(0, 1);
  sys->log<System::MESSAGE>("A normally distributed number with mean 0 and stdev 1: %.13g", normal);

  //System also allows to access the argc and argv passed at construction
  {
    auto argc = sys->getargc();
    auto argv = sys->getargv();
    sys->log<System::MESSAGE>("The name of this executable is %s, %d arguments were passed to it.", argv[0], argc-1);
  }

  //GPU memory allocation is really slow, for that matter System provides a C++ compatible GPU memory pool allocator.
  // This allocator caches queries to it, so multiple allocations/deallocations of similar sizes will be almost instantaneous
  // We will just mention it exists or now, though, and leave it to the advanced examples.
  // auto alloc =  sys->getTemporaryDeviceAllocator<double>();
  // thrust::device_vector<double, System::allocator_thrust<double>> vec(10000, alloc);

  //Destroy the UAMMD environment and exit
  sys->finish();
  return 0;
}
