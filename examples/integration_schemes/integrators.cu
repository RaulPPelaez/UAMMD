#include "hip/hip_runtime.h"
/* Raul P. Pelaez 2021

   This code contains a collection of functions that create and return an instance of the different UAMMD integrators.
   The code in itself does not do much, rather it is intended  to serve as a copy pastable collection of snippets.

 */

#include<uammd.cuh>


using namespace uammd;

//I like to place these basic UAMMD objects in a struct so it is easy to pass them around
struct UAMMD{
  std::shared_ptr<ParticleData> pd;
  std::shared_ptr<System> sys;
  //Parameters par; //For this example parameters will be hardcoded
};

//Creates and returns a UAMMD struct with the basics that we have covered in previous tutorials
UAMMD initializeUAMMD(int argc, char *argv[], int numberParticles){
  UAMMD sim;
  //Initialize System and ParticleData
  sim.sys = std::make_shared<System>(argc, argv);
  sim.pd = std::make_shared<ParticleData>(sim.sys, numberParticles);
  return sim;
}

#include<Integrator/BrownianDynamics.cuh>
//There are several BD integrators
using BDMethod = BD::EulerMaruyama;
// using BDMethod = BD::MidPoint;
// using BDMethod = BD::AdamsBashforth;
// using BDMethod = BD::Leimkuhler;
std::shared_ptr<Integrator> createIntegratorBD(UAMMD sim){
  typename BDMethod::Parameters par;
  par.temperature = 1.0;
  par.viscosity = 1.0;
  par.hydrodynamicRadius = 1.0; //Self diffusion coefficient will be D = T*M = T/(6*pi*vis*hydrodynamicRadius)
  par.dt = 0.001;
  //Optionally you can place a shear matrix, dX = M*F*dt + sqrt(2*D*dt)*dW + K*R
  //par.K = {{1,2,3},{1,2,3},{1,2,3}};
  //or
  //par.K[0] = {1,2,3};
  //or
  //par.K[1].x = 1;
  //All K elements start being zero.
  auto pg = std::make_shared<ParticleGroup>(sim.pd, sim.sys, "All");
  return std::make_shared<BDMethod>(sim.pd, pg, sim.sys, par);
}

#include "Integrator/VerletNVT.cuh"
using Verlet = VerletNVT::GronbechJensen;
//using Verlet = VerletNVT::Basic; //A velocity reescaling algorithm
std::shared_ptr<Integrator> createIntegratorVerletNVT(UAMMD sim){
  typename Verlet::Parameters par;
  par.temperature = 1.0;
  par.friction = 1.0;
  par.dt = 0.1;
  //If present, all particles will have this mass, otherwise the individual particle masses in ParticleData will be used
  //If those masses have not been set then the default mass is 1.0.
  //par.mass = 1.0;
  //If set to false particle velocities will be left untouched during initialization
  //If true (default) velocities will be sampled from the equilibrium configuration
  //par.initVelocities = false;
  auto pg = std::make_shared<ParticleGroup>(sim.pd, sim.sys, "All");
  return std::make_shared<Verlet>(sim.pd, pg, sim.sys, par);
}

#include "Integrator/VerletNVE.cuh"
std::shared_ptr<Integrator> createIntegratorVerletNVE(UAMMD sim){
  typename VerletNVE::Parameters par;
  par.dt = 0.1;
  par.energy = 1; //Optionally a target energy can be passed that VerletNVE will set according to velocities keep constant
  //par.initVelocities = false; //If true, velocities will be initialized by the module to ensure the desired energy
  //Note that it does not make sense to pass an energy and prevent VerletNVE from initializing velocities to match it.
  //If present, all particles will have this mass, otherwise the individual particle masses in ParticleData will be used
  //If those masses have not been set then the default mass is 1.0.
  //par.mass = 1.0;
  auto pg = std::make_shared<ParticleGroup>(sim.pd, sim.sys, "All");
  return std::make_shared<VerletNVE>(sim.pd, pg, sim.sys, par);
}

#include "Integrator/VerletNVE.cuh"
#include "Interactor/Potential/DPD.cuh"
#include"Interactor/PairForces.cuh"
//DPD is handled by UAMMD as a VerletNVE integrator with a special short range interaction
std::shared_ptr<Integrator> createIntegratorDPD(UAMMD sim){
  using NVE = VerletNVE;
  NVE::Parameters par;
  par.dt = 1.0;
  par.initVelocities = false;
  auto pg = std::make_shared<ParticleGroup>(sim.pd, sim.sys, "All");
  auto verlet = std::make_shared<NVE>(sim.pd, pg, sim.sys, par);
  using DPD = PairForces<Potential::DPD>;
  Potential::DPD::Parameters dpd_params;
  dpd_params.cutOff = 1.0;
  dpd_params.temperature = 1.0;
  dpd_params.gamma = 1.0;
  dpd_params.A = 1.0;
  dpd_params.dt = 0.1;
  auto pot = std::make_shared<Potential::DPD>(sim.sys, dpd_params);
  DPD::Parameters params;
  real3 L = make_real3(32,32,32);
  params.box = Box(L);
  auto pairforces = std::make_shared<DPD>(sim.pd, pg, sim.sys, params, pot);
  verlet->addInteractor(pairforces);
  return verlet;
}

#include "Integrator/VerletNVE.cuh"
#include "Interactor/SPH.cuh"
//SPH is handled by UAMMD as a VerletNVE integrator with a special interaction
std::shared_ptr<Integrator> createIntegratorSPH(UAMMD sim){
  using NVE = VerletNVE;
  NVE::Parameters par;
  par.dt = 0.1;
  par.initVelocities = false;
  auto pg = std::make_shared<ParticleGroup>(sim.pd, sim.sys, "All");
  auto verlet = std::make_shared<NVE>(sim.pd, pg, sim.sys, par);
  SPH::Parameters params;
  real3 L = make_real3(32,32,32);
  params.box = Box(L);
  //Pressure for a given particle "i" in SPH will be computed as gasStiffness·(density_i - restDensity)
  //Where density is computed as a function of the masses of the surroinding particles
  //Particle mass starts as 1, but you can change this in customizations.cuh
  params.support = 2.4;   //Cut off distance for the SPH kernel
  params.viscosity = 1.0;   //Environment viscosity
  params.gasStiffness = 1.0;
  params.restDensity = 1.0;
  auto sph = std::make_shared<SPH>(sim.pd, pg, sim.sys, params);
  verlet->addInteractor(sph);
  return verlet;
}

#include"Integrator/BDHI/BDHI_EulerMaruyama.cuh"
#include"Integrator/BDHI/BDHI_PSE.cuh"
#include"Integrator/BDHI/BDHI_FCM.cuh"
//Creates a triply periodic Brownian Dynamics with Hydrodynamic Interactions integration module
std::shared_ptr<Integrator> createIntegratorBDHI(UAMMD sim){
  //There are several hydrodynamics modules, we choose between Positively Split Ewald (PSE) or Force Coupling Method (FCM) here
  // mainly for performance reasons. FCM is faster for small and/or dense systems, but it is limited in the system size by memory.
  // PSE can be slower it temperature>0, but does not have that system size constraints.
  //FCM scales linearly with system size (so doubling the box size in the three dimensions makes it 8 times slower) and number of particles
  //PSE scales linearly with the number of particles, independently of system size. But the "psi" parameter must be tweaked to find the optimal performance for each case.
  //See the wiki for more information about these modules
  real3 L = make_real3(32,32,32);
  real hydrodynamicRadius = 1.0;
  real maxL = std::max({L.x, L.y, L.z});
  int maxcells = maxL/hydrodynamicRadius;
  //In both modules, particle self diffusion coefficient will be T/(6*pi*viscosity*hydrodynamicRadius) or close to it
  if(maxcells >= 128){
    using Scheme = BDHI::PSE;
    Scheme::Parameters par;
    par.box = Box(L);
    par.temperature = 1.0;
    par.viscosity = 1.0;
    par.dt = 0.1;
    par.hydrodynamicRadius = hydrodynamicRadius;
    par.tolerance = 1e-4;
    //Balances the load of the algorithm, low values work best for dilute and/or big systems.
    // Higher values will work best for dense and/or small systems.
    par.psi = 1.0/par.hydrodynamicRadius;
    auto bdhi = std::make_shared<BDHI::EulerMaruyama<Scheme>>(sim.pd, sim.sys, par);
    return bdhi;
  }
  else{
    using Scheme = BDHI::FCM;
    Scheme::Parameters par;
    par.box = Box(L);
    par.temperature = 1.0;
    par.viscosity = 1.0;
    par.dt = 0.1;
    par.hydrodynamicRadius = hydrodynamicRadius;
    par.tolerance = 1e-4;
    auto bdhi = std::make_shared<BDHI::EulerMaruyama<Scheme>>(sim.pd, sim.sys, par);
    return bdhi;
  }
}


std::shared_ptr<Integrator> createIntegratorFCM(UAMMD sim){
  //The FCM module also works as an standalone Integrator.
  //In this mode, FCM can also compute angular displacements due to torques acting on the particles
  //See the wiki for more information about these modules
  real3 L = make_real3(32,32,32);
  real hydrodynamicRadius = 1.0;
  BDHI::FCMIntegrator::Parameters par;
  par.box = Box(L);
  par.temperature = 1.0;
  par.viscosity = 1.0;
  par.dt = 0.1;
  par.hydrodynamicRadius = hydrodynamicRadius;
  par.tolerance = 1e-4;
  auto bdhi = std::make_shared<BDHI::FCMIntegrator>(sim.pd, par);
  return bdhi;
}
int main(int argc, char* argv[]){
  int N = 16384;
  auto sim = initializeUAMMD(argc, argv, N);
  auto integrator = createIntegratorBD(sim);
  for(int i= 0; i<100; i++){
    integrator->forwardTime();
  }
  sim.sys->finish();
  return 0;
}
